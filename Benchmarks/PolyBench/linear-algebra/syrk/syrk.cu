#include "hip/hip_runtime.h"
/**
 * syrk.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "syrk.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU


void init_arrays(int ni, int nj,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(C,NI,NI,ni,ni),
		DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj))
{
	int i, j;

	*alpha = 32412;
	*beta = 2123;
	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nj; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / ni;
		}
	}

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < ni; j++)
		{
			C[i][j] = ((DATA_TYPE) i*j) / ni;
		}
	}
}


void syrk(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni))
{
	int i, j, k;
	
	/*  C := alpha*A*A' + beta*C */
	for (i = 0; i < _PB_NI; i++)
	{
		for (j = 0; j < _PB_NI; j++)
		{
			C[i][j] *= beta;
		}
	}
	
	for (i = 0; i < _PB_NI; i++)
	{
		for (j = 0; j < _PB_NI; j++)
		{
			for (k = 0; k < _PB_NJ; k++)
			{
				C[i][j] += alpha * A[i][k] * A[j][k];
			}
		}
	}
}


void compareResults(int ni, DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni), DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NI, ni, ni))
{
	int i,j,fail;
	fail = 0;

	// Compare C with D
	for (i=0; i<ni; i++)
	{
		for (j=0; j<ni; j++)
		{
			if (percentDiff(C[i][j], C_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
	
	return;
}


__global__ void syrk_kernel(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *c)
{
	/*  C := alpha*A*A' + beta*C */
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NI))
	{
		c[i * NI + j] *= beta;
		int k;		
		for(k=0; k < _PB_NJ; k++)
		{
			c[i * NI + j] += alpha * a[i * NJ + k] * a[j * NJ + k];
		}
	}
}


void syrkCuda(int ni, int nj, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C, NI, NI, ni, ni), 
		DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NI, ni, ni))
{
	DATA_TYPE* A_gpu;
	DATA_TYPE* C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NI);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NI, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil(((float)NI) / ((float)DIM_THREAD_BLOCK_X))), (size_t)ceil(((float)NI) / ((float)DIM_THREAD_BLOCK_Y)));

	/* Start timer. */
  	polybench_start_instruments;

	syrk_kernel<<<grid,block>>>(ni, nj, alpha, beta, A_gpu,C_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NI, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(C_gpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
// static
// void print_array(int ni, DATA_TYPE POLYBENCH_2D(C,NI,NI,ni,ni))
// {
//   int i, j;

//   for (i = 0; i < ni; i++)
//     for (j = 0; j < ni; j++) {
// 	fprintf (stderr, DATA_PRINTF_MODIFIER, C[i][j]);
// 	if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
//     }
//   fprintf (stderr, "\n");
// }


int main(int argc, char *argv[])
{
	/* Retrieve problem size. */
	int ni = NI;
	int nj = NJ;

	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NJ,ni,nj);
  	POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NI,ni,ni);
  	POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu,DATA_TYPE,NI,NI,ni,ni);

	init_arrays(ni, nj, &alpha, &beta, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(A));
	
	GPU_argv_init();	
	syrkCuda(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

	// #ifdef RUN_ON_CPU

	// 	/* Start timer. */
	//   	polybench_start_instruments;

	// 	syrk(ni, nj, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(C));

	// 	/* Stop and print timer. */
	// 	printf("CPU Time in seconds:\n");
  	// 	polybench_stop_instruments;
 	// 	polybench_print_instruments;

	// 	compareResults(ni, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

	// #else //prevent dead code elimination

	// 	polybench_prevent_dce(print_array(ni, POLYBENCH_ARRAY(C_outputFromGpu)));

	// #endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
  	POLYBENCH_FREE_ARRAY(C);
	POLYBENCH_FREE_ARRAY(C_outputFromGpu);

	return 0;
}

#include <polybench.c>