#include "hip/hip_runtime.h"
/**
 * lu.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>

#define POLYBENCH_TIME 1

#include "lu.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

#define RUN_ON_CPU


void lu(int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
	for (int k = 0; k < _PB_N; k++)
    {
		for (int j = k + 1; j < _PB_N; j++)
		{
			A[k][j] = A[k][j] / A[k][k];
		}

		for (int i = k + 1; i < _PB_N; i++)
		{
			for (int j = k + 1; j < _PB_N; j++)
			{
				A[i][j] = A[i][j] - A[i][k] * A[k][j];
			}
		}
    }
}


void init_array(int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
	int i, j;

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j + 1) / N;
		}
	}
}


void compareResults(int n, DATA_TYPE POLYBENCH_2D(A_cpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n))
{
	int i, j, fail;
	fail = 0;
	
	// Compare a and b
	for (i=0; i<n; i++) 
	{
		for (j=0; j<n; j++) 
		{
			if (percentDiff(A_cpu[i][j], A_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void lu_kernel1(int n, DATA_TYPE *A, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((j > k) && (j < _PB_N))
	{
		A[k*N + j] = A[k*N + j] / A[k*N + k];
	}
}


__global__ void lu_kernel2(int n, DATA_TYPE *A, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i > k) && (j > k) && (i < _PB_N) && (j < _PB_N))
	{
		A[i*N + j] = A[i*N + j] - A[i*N + k] * A[k*N + j];
	}
}


void luCuda(int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n))
{
	DATA_TYPE* AGpu;

	hipMalloc(&AGpu, N * N * sizeof(DATA_TYPE));
	hipMemcpy(AGpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid1(1, 1, 1);
	dim3 grid2(1, 1, 1);

	/* Start timer. */
  	polybench_start_instruments;

	for (int k = 0; k < N; k++)
	{
		grid1.x = (unsigned int)(ceil((float)(N - (k + 1)) / ((float)block1.x)));
		lu_kernel1<<<grid1, block1>>>(n, AGpu, k);
		hipDeviceSynchronize();

		grid2.x = (unsigned int)(ceil((float)(N - (k + 1)) / ((float)block2.x)));
		grid2.y = (unsigned int)(ceil((float)(N - (k + 1)) / ((float)block2.y)));
		lu_kernel2<<<grid2, block2>>>(n, AGpu, k);
		hipDeviceSynchronize();
	}
	
	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(A_outputFromGpu, AGpu, N * N * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
	hipFree(AGpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
// static
// void print_array(int n,
// 		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))

// {
//   int i, j;

//   for (i = 0; i < n; i++)
//     for (j = 0; j < n; j++) {
//       fprintf (stderr, DATA_PRINTF_MODIFIER, A[i][j]);
//       if ((i * n + j) % 20 == 0) fprintf (stderr, "\n");
//     }
//   fprintf (stderr, "\n");
// }
	

int main(int argc, char *argv[])
{
	int n = N;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,N,N,n,n);
  	POLYBENCH_2D_ARRAY_DECL(A_outputFromGpu,DATA_TYPE,N,N,n,n);

	init_array(n, POLYBENCH_ARRAY(A));

	GPU_argv_init();
	luCuda(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_outputFromGpu));
	

	// #ifdef RUN_ON_CPU

	// 	/* Start timer. */
	//   	polybench_start_instruments;

	// 	lu(n, POLYBENCH_ARRAY(A));

	// 	/* Stop and print timer. */
	// 	printf("CPU Time in seconds:\n");
	//   	polybench_stop_instruments;
	//  	polybench_print_instruments;
	
	// 	compareResults(n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_outputFromGpu));

	// #else //prevent dead code elimination

	// 	polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A_outputFromGpu)));

	// #endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(A_outputFromGpu);

   	return 0;
}

#include <polybench.c>