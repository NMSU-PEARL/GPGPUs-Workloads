#include "hip/hip_runtime.h"
/**
 * 3DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "convolution3D.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

#define RUN_ON_CPU


void conv3D(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk))
{
	int i, j, k;
	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;

	for (i = 1; i < _PB_NI - 1; ++i) // 0
	{
		for (j = 1; j < _PB_NJ - 1; ++j) // 1
		{
			for (k = 1; k < _PB_NK -1; ++k) // 2
			{
				B[i][j][k] = c11 * A[(i - 1)][(j - 1)][(k - 1)]  +  c13 * A[(i + 1)][(j - 1)][(k - 1)]
					     +   c21 * A[(i - 1)][(j - 1)][(k - 1)]  +  c23 * A[(i + 1)][(j - 1)][(k - 1)]
					     +   c31 * A[(i - 1)][(j - 1)][(k - 1)]  +  c33 * A[(i + 1)][(j - 1)][(k - 1)]
					     +   c12 * A[(i + 0)][(j - 1)][(k + 0)]  +  c22 * A[(i + 0)][(j + 0)][(k + 0)]   
					     +   c32 * A[(i + 0)][(j + 1)][(k + 0)]  +  c11 * A[(i - 1)][(j - 1)][(k + 1)]  
					     +   c13 * A[(i + 1)][(j - 1)][(k + 1)]  +  c21 * A[(i - 1)][(j + 0)][(k + 1)]  
					     +   c23 * A[(i + 1)][(j + 0)][(k + 1)]  +  c31 * A[(i - 1)][(j + 1)][(k + 1)]  
					     +   c33 * A[(i + 1)][(j + 1)][(k + 1)];
			}
		}
	}
}


void init(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk))
{
	int i, j, k;

	for (i = 0; i < ni; ++i)
    	{
		for (j = 0; j < nj; ++j)
		{
			for (k = 0; k < nk; ++k)
			{
				A[i][j][k] = i % 12 + 2 * (j % 7) + 3 * (k % 13);
			}
		}
	}
}


void compareResults(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B_outputFromGpu, NI, NJ, NK, ni, nj, nk))
{
	int i, j, k, fail;
	fail = 0;
	
	// Compare result from cpu and gpu
	for (i = 1; i < ni - 1; ++i) // 0
	{
		for (j = 1; j < nj - 1; ++j) // 1
		{
			for (k = 1; k < nk - 1; ++k) // 2
			{
				if (percentDiff(B[i][j][k], B_outputFromGpu[i][j][k]) > PERCENT_DIFF_ERROR_THRESHOLD)
				{
					fail++;
				}
			}	
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void convolution3D_kernel(int ni, int nj, int nk, DATA_TYPE* A, DATA_TYPE* B, int i)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;


	if ((i < (_PB_NI-1)) && (j < (_PB_NJ-1)) &&  (k < (_PB_NK-1)) && (i > 0) && (j > 0) && (k > 0))
	{
		B[i*(NK * NJ) + j*NK + k] = c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c21 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c23 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c31 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c33 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c12 * A[(i + 0)*(NK * NJ) + (j - 1)*NK + (k + 0)]  +  c22 * A[(i + 0)*(NK * NJ) + (j + 0)*NK + (k + 0)]   
					     +   c32 * A[(i + 0)*(NK * NJ) + (j + 1)*NK + (k + 0)]  +  c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k + 1)]  
					     +   c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k + 1)]  +  c21 * A[(i - 1)*(NK * NJ) + (j + 0)*NK + (k + 1)]  
					     +   c23 * A[(i + 1)*(NK * NJ) + (j + 0)*NK + (k + 1)]  +  c31 * A[(i - 1)*(NK * NJ) + (j + 1)*NK + (k + 1)]  
					     +   c33 * A[(i + 1)*(NK * NJ) + (j + 1)*NK + (k + 1)];
	}
}


void convolution3DCuda(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B_outputFromGpu, NI, NJ, NK, ni, nj, nk))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ * NK, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil( ((float)NK) / ((float)block.x) )), (size_t)(ceil( ((float)NJ) / ((float)block.y) )));
	
	/* Start timer. */
  	polybench_start_instruments;

	int i;
	for (i = 1; i < _PB_NI - 1; ++i) // 0
	{
		convolution3D_kernel<<< grid, block >>>(ni, nj, nk, A_gpu, B_gpu, i);
	}

	hipDeviceSynchronize();
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;
	
	hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK, hipMemcpyDeviceToHost);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
}


// /* DCE code. Must scan the entire live-out data.
//    Can be used also to check the correctness of the output. */
// static
// void print_array(int ni, int nj, int nk,
// 		 DATA_TYPE POLYBENCH_3D(B,NI,NJ,NK,ni,nj,nk))
// {
//   int i, j, k;

//   for (i = 0; i < ni; i++)
//     for (j = 0; j < nj; j++) 
// 	for (k = 0; k < nk; k++)
// 	{
// 	fprintf (stderr, DATA_PRINTF_MODIFIER, B[i][j][k]);
// 	if ((i * (nj*nk) + j*nk + k) % 20 == 0) fprintf (stderr, "\n");
//     }
//   fprintf (stderr, "\n");
// }


int main(int argc, char *argv[])
{
	int ni = NI;
	int nj = NJ;
	int nk = NK;

	POLYBENCH_3D_ARRAY_DECL(A,DATA_TYPE,NI,NJ,NK,ni,nj,nk);
	POLYBENCH_3D_ARRAY_DECL(B,DATA_TYPE,NI,NJ,NK,ni,nj,nk);
	POLYBENCH_3D_ARRAY_DECL(B_outputFromGpu,DATA_TYPE,NI,NJ,NK,ni,nj,nk);

	init(ni, nj, nk, POLYBENCH_ARRAY(A));
	
	GPU_argv_init();

	convolution3DCuda(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

	// #ifdef RUN_ON_CPU

	// 	/* Start timer. */
	//   	polybench_start_instruments;

	// 	conv3D(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

	// 	printf("CPU Time in seconds:\n");
	//   	polybench_stop_instruments;
	//  	polybench_print_instruments;
	
	// 	compareResults(ni, nj, nk, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

	// #else //prevent dead code elimination

	// 	polybench_prevent_dce(print_array(NI, NJ, NK, POLYBENCH_ARRAY(B_outputFromGpu)));

	// #endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(B_outputFromGpu);

    	return 0;
}

#include <polybench.c>